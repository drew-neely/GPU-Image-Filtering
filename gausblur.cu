#include "hip/hip_runtime.h"
#include <math.h>

#include "image.h"
#include "filter.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


double GausBlur::kernelAt(double strength, int x, int y) {
	return 1.0 / (2 * M_PI * strength * strength) *
		exp(-1.0 * (x*x + y*y) / (2.0 * strength * strength));
}

__host__ __device__
int GausBlur::getIndex(int x, int y) {
	return y * size + x;
}

// strength = stdev in gaussian transformation equation
// radius == -1 => choose the radius to make it such that an increase in
//		radius of 1 sees an inclusion increase of < 1%
GausBlur::GausBlur(double strength, int radius) : Filter(1) {
	if(radius == -1) {
		double loss;
		do {
			radius++;
			loss = 0;
			for(int i = 1; i < radius; i++) {
				loss += kernelAt(strength, i, radius) * 8;
			}
			loss += kernelAt(strength, radius, 0) * 4;
			loss += kernelAt(strength, radius, radius) * 4;
		} while(loss > 0.01);
		radius --;
		this->radius = radius;
		this->size = radius * 2 + 1;
	} else {
		this->radius = radius;
		this->size = radius * 2 + 1;
	}
	printf("size %d\n", size);

	// host_vector<double> kernelHV = host_vector<double>(size*size);
	kernel = (double*) malloc(sizeof(double) * size * size);

	for(int y = 0; y <= radius; y++) {
		for(int x = 0; x <= radius; x++) {
			double val = kernelAt(strength, x, y);
			// kernelHV[getIndex(radius + y, radius + x)] = val;
			// kernelHV[getIndex(radius + y, radius - x)] = val;
			// kernelHV[getIndex(radius - y, radius + x)] = val;
			// kernelHV[getIndex(radius - y, radius - x)] = val;
			kernel[getIndex(radius + y, radius + x)] = val;
			kernel[getIndex(radius + y, radius - x)] = val;
			kernel[getIndex(radius - y, radius + x)] = val;
			kernel[getIndex(radius - y, radius - x)] = val;
		}
	}
	
	// this.kernel = (double*) kernelHV.data();
	// device_vector<double> kernelDV = kernelHV;
	// printf("size: %d\n", size);
	// printf("kernel[27] = %f\n",   (double)kernel[27]);
	// printf("kernelHV[27] = %f\n", (double)kernelHV[27]);
	// printf("kernelDV[27] = %f\n", (double)kernelDV[27]);
	// this.kernelDev = kernelDV.data();
	// printf("kernelDeV[27] = %f\n", (double)this.kernelDeV[27]);
	// printf("kernelDeV = %p\n", this.kernelDeV);

}

__host__
Pixel GausBlur::transform(Pixel* data, int width, int height, int cx, int cy) {
	// bool v = (cx == 100 && cy == 100) || (cx == 105 && cy == 100) || (cx == 100 && cy == 99); 
	// if(v) printf("(%d, %d)\n", cx, cy);
	double r = 0;
	double g = 0;
	double b = 0;
	double totalWeight = 0;
	for(int y = cy - radius; y < cy + radius + 1; y++) {
		for(int x = cx - radius; x < cx + radius + 1; x++) {
			if(y >= 0 && y < height && x >= 0 && x < width) {
				Pixel p = data[width * y + x];
				double k = kernel[getIndex(y - (cy - radius), x - (cx - radius))];
				// if(v) printf("(%d, %d) => %f\n", x, y, k);
				r += p.r * k;
				g += p.g * k;
				b += p.b * k;
				totalWeight += k;
			}
		}
	}
	r /= totalWeight; // scale for edges and rounded kernel
	g /= totalWeight;
	b /= totalWeight;
	Pixel newPixel = Pixel(round(r), round(g), round(b));
	return newPixel;

}

__host__ __device__
Pixel GausBlur::transform(device_ptr<Pixel> data, device_ptr<double> kernel, int width, int height, int cx, int cy) {
	// bool v = (cx == 100 && cy == 100) || (cx == 105 && cy == 100) || (cx == 100 && cy == 99); 
	// if(v) printf("kernel = %p\n", (void*) kernel);
	// if(v) printf("(%d, %d)\n", cx, cy);
	double r = 0;
	double g = 0;
	double b = 0;
	double totalWeight = 0;
	for(int y = cy - radius; y < cy + radius + 1; y++) {
		for(int x = cx - radius; x < cx + radius + 1; x++) {
			if(y >= 0 && y < height && x >= 0 && x < width) {
				Pixel p = data[width * y + x];
				double k = kernel[getIndex(y - (cy - radius), x - (cx - radius))];
				// if(v) printf("(%d, %d) => %f\n", x, y, k);
				r += p.r * k;
				g += p.g * k;
				b += p.b * k;
				totalWeight += k;
			}
		}
	}
	r /= totalWeight; // scale for edges and rounded kernel
	g /= totalWeight;
	b /= totalWeight;
	Pixel newPixel = Pixel(round(r), round(g), round(b));
	return newPixel;

}